
#include <hip/hip_runtime.h>
#include <fstream>
using namespace std;

////////////////////////////////////////////////////////////////////////////////
// Process an array of optN options on CPU
////////////////////////////////////////////////////////////////////////////////
extern "C" void BlackScholesCPU(
    float *h_CallResult,
    float *h_PutResult,
    float *h_StockPrice,
    float *h_OptionStrike,
    float *h_OptionYears,
    float Riskfree,
    float Volatility,
    int optN
);

///////////////////////////////////////////////////////////////////////////////
// Polynomial approximation of cumulative normal distribution function
///////////////////////////////////////////////////////////////////////////////
__device__ inline float cndGPU(float d) {
    const float       A1 = 0.31938153f;
    const float       A2 = -0.356563782f;
    const float       A3 = 1.781477937f;
    const float       A4 = -1.821255978f;
    const float       A5 = 1.330274429f;
    const float RSQRT2PI = 0.39894228040143267793994605993438f;

    float
    K = 1.0f / (1.0f + 0.2316419f * fabsf(d));

    float
    cnd = RSQRT2PI * __expf(- 0.5f * d * d) *
          (K * (A1 + K * (A2 + K * (A3 + K * (A4 + K * A5)))));

    if (d > 0)
        cnd = 1.0f - cnd;

    return cnd;
}


///////////////////////////////////////////////////////////////////////////////
// Black-Scholes formula for both call and put
///////////////////////////////////////////////////////////////////////////////
__device__ inline void BlackScholesBodyGPU(
    float &CallResult,
    float &PutResult,
    float S, //Stock price
    float X, //Option strike
    float T, //Option years
    float R, //Riskless rate
    float V  //Volatility rate
) {
    float sqrtT, expRT;
    float d1, d2, CNDD1, CNDD2;

    float parrotOutput[1];
    
    sqrtT = sqrtf(T);
    d1 = (__logf(S / X) + (R + 0.5f * V * V) * T) / (V * sqrtT);
    d2 = d1 - V * sqrtT;

    CNDD1 = cndGPU(d1);
    CNDD2 = cndGPU(d2);

    // Calculate Call and Put simultaneously
    expRT = __expf(- R * T);
    CallResult = S * CNDD1 - X * expRT * CNDD2;
    parrotOutput[0] = CallResult / 10.0;

    CallResult = parrotOutput[0] * 10.0;
    PutResult  = X * expRT * (1.0f - CNDD2) - S * (1.0f - CNDD1);
}


////////////////////////////////////////////////////////////////////////////////
//Process an array of optN options on GPU
////////////////////////////////////////////////////////////////////////////////
__global__ void BlackScholesGPU(
    float *d_CallResult,
    float *d_PutResult,
    float *d_StockPrice,
    float *d_OptionStrike,
    float *d_OptionYears,
    float Riskfree,
    float Volatility,
    int optN
) {
    const int opt = blockDim.x * blockIdx.x + threadIdx.x;
    if (opt < optN)
        BlackScholesBodyGPU(
            d_CallResult[opt],
            d_PutResult[opt],
            d_StockPrice[opt],
            d_OptionStrike[opt],
            d_OptionYears[opt],
            Riskfree,
            Volatility
        );
}

////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
const int OPT_N = 4000000;
const int  NUM_ITERATIONS = 1;

const int   OPT_SZ      = OPT_N * sizeof(float);
const float RISKFREE    = 0.02f;
const float VOLATILITY  = 0.30f;

#define DIV_UP(a, b) ( ((a) + (b) - 1) / (b) )

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {

    if (argc < 3) {
        printf("Usage: %s <input-file> <output-file>\n", argv[0]);
        exit(1);
    }

    char *inputFilename = argv[1];
    char *outputFilename = argv[2];

    // ======================================
    // == Declaring variables
    // ======================================
    // > Host data
    float *h_CallResultGPU, *h_PutResultGPU; // CPU copy of GPU results
    float *h_StockPrice, *h_OptionStrike, *h_OptionYears; // CPU instance of input data
    
    // > Device data
    float *d_CallResult, *d_PutResult; // Results calculated by GPU
    float *d_StockPrice, *d_OptionStrike, *d_OptionYears; // GPU instance of input data
    
    int i;

    // ======================================
    // == Allocating memory
    // ======================================
    // > Host data
    h_CallResultGPU = (float *)malloc(OPT_SZ);
    h_PutResultGPU  = (float *)malloc(OPT_SZ);
    h_StockPrice    = (float *)malloc(OPT_SZ);
    h_OptionStrike  = (float *)malloc(OPT_SZ);
    h_OptionYears   = (float *)malloc(OPT_SZ);

    // > Device data
    hipMalloc((void **)&d_CallResult,   OPT_SZ);
    hipMalloc((void **)&d_PutResult,    OPT_SZ);
    hipMalloc((void **)&d_StockPrice,   OPT_SZ);
    hipMalloc((void **)&d_OptionStrike, OPT_SZ);
    hipMalloc((void **)&d_OptionYears,  OPT_SZ);

    // ======================================
    // == Reading input data
    // ======================================
    std::ifstream dataFile(inputFilename);

    int numberOptions;
    dataFile >> numberOptions;

    float stockPrice, optionStrike, optionYear;
    for (i = 0; i < numberOptions; i++) {
        dataFile >> stockPrice >> optionStrike >> optionYear;
        h_StockPrice[i] = stockPrice;
        h_OptionStrike[i] = optionStrike;
        h_OptionYears[i] =  optionYear;      
    }

    // ======================================
    // == Copying data to device
    // ======================================
    hipMemcpy(d_StockPrice,    h_StockPrice,   numberOptions * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_OptionStrike,  h_OptionStrike, numberOptions * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_OptionYears,   h_OptionYears,  numberOptions * sizeof(float), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    // ======================================
    // == Executing on device
    // ======================================
    for (i = 0; i < NUM_ITERATIONS; i++) {
        BlackScholesGPU<<<DIV_UP(numberOptions, 128), 128>>>(
            d_CallResult,
            d_PutResult,
            d_StockPrice,
            d_OptionStrike,
            d_OptionYears,
            RISKFREE,
            VOLATILITY,
            numberOptions
        ); 
    }

    hipDeviceSynchronize();

    // ======================================
    // == Reading back results from device
    // ======================================
    hipMemcpy(h_CallResultGPU, d_CallResult, numberOptions * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_PutResultGPU,  d_PutResult,  numberOptions * sizeof(float), hipMemcpyDeviceToHost);

    // ======================================
    // == Writing results to output file
    // ======================================
    ofstream callResultFile;
    callResultFile.open(outputFilename);
    for (i = 0 ; i < numberOptions; i++) {
        callResultFile << h_CallResultGPU[i] << std::endl;
    }
    callResultFile.close();

    // ======================================
    // == Deallocating memory
    // ======================================
    hipFree(d_OptionYears);
    hipFree(d_OptionStrike);
    hipFree(d_StockPrice);
    hipFree(d_PutResult);
    hipFree(d_CallResult);
    free(h_OptionYears);
    free(h_OptionStrike);
    free(h_StockPrice);
    free(h_PutResultGPU);
    free(h_CallResultGPU);

    hipDeviceReset();
    exit(EXIT_SUCCESS);
}
