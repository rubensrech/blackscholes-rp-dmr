#include "hip/hip_runtime.h"
#include "util.h"
#include "kernels.h"
#include "dmr-constants.h"

////////////////////////////////////////////////////////////////////////////////
// Check error functions
////////////////////////////////////////////////////////////////////////////////

__device__ unsigned long long errors = 0;

__forceinline__  __device__ void relativeError(double rhs, float lhs, float REL_ERR_THRESHOLD) {
    float relErr = abs(1 - lhs / float(rhs));
    if (relErr > REL_ERR_THRESHOLD) {
        atomicAdd(&errors, 1);
    }
}

__forceinline__  __device__ void uintError(double rhs, float lhs, float UINT_ERR_THRESHOLD) {
	float rhs_as_float = float(rhs);
	uint32_t lhs_data = *((uint32_t*) &lhs);
	uint32_t rhs_data = *((uint32_t*) &rhs_as_float);

	uint32_t uintErr = SUB_ABS(lhs_data, rhs_data);

	if (uintErr > UINT_ERR_THRESHOLD) {
		atomicAdd(&errors, 1);
	}
}

__device__ void checkErrors(double rhs, float lhs, float THRESHOLD) {
#if ERROR_METRIC == UINT_ERROR
    uintError(rhs, lhs, THRESHOLD);
#else
    relativeError(rhs, lhs, THRESHOLD);
#endif
}

// > Getters

unsigned long long getDMRErrors() {
    unsigned long long ret = 0;
    hipMemcpyFromSymbol(&ret, HIP_SYMBOL(errors), sizeof(unsigned long long), 0, hipMemcpyDeviceToHost);
    return ret;
}

////////////////////////////////////////////////////////////////////////////////
// BLACK SCHOLES
////////////////////////////////////////////////////////////////////////////////

const double RISKFREE    = 0.02f;
const double VOLATILITY  = 0.30f;

__device__ inline double cndGPU(double d) {
    const double       A1 = 0.31938153f;
    const double       A2 = -0.356563782f;
    const double       A3 = 1.781477937f;
    const double       A4 = -1.821255978f;
    const double       A5 = 1.330274429f;
    const double RSQRT2PI = 0.39894228040143267793994605993438f;

    double K = 1.0f / (1.0f + 0.2316419f * fabs(d));

    double cnd = RSQRT2PI * exp(- 0.5f * d * d) *
            (K * (A1 + K * (A2 + K * (A3 + K * (A4 + K * A5)))));

    if (d > 0)
        cnd = 1.0f - cnd;

    return cnd;
}

__device__ inline float cndGPU(float d) {
    const float       A1 = 0.31938153f;
    const float       A2 = -0.356563782f;
    const float       A3 = 1.781477937f;
    const float       A4 = -1.821255978f;
    const float       A5 = 1.330274429f;
    const float RSQRT2PI = 0.39894228040143267793994605993438f;

    float K = 1.0f / (1.0f + 0.2316419f * fabsf(d));

    float cnd = RSQRT2PI * __expf(- 0.5f * d * d) *
            (K * (A1 + K * (A2 + K * (A3 + K * (A4 + K * A5)))));

    if (d > 0)
        cnd = 1.0f - cnd;

    return cnd;
}

__device__ inline void BlackScholesBodyGPU(double &CallResult, double &PutResult, float &CallResult_rp,
        float &PutResult_rp, double S, double X, double T, double R, double V) {
    // > Full-precision
    double sqrtT, expRT;
    double d1, d2, CNDD1, CNDD2;
    // > Reduced-precision
    float S_rp = float(S), X_rp = float(X), T_rp = float(T), R_rp = float(R), V_rp = float(V);
    float sqrtT_rp, expRT_rp;
    float d1_rp, d2_rp, CNDD1_rp, CNDD2_rp;
    
    // > Full-precision
    sqrtT = sqrt(T);
    d1 = (log(S / X) + (R + 0.5f * V * V) * T) / (V * sqrtT);
    d2 = d1 - V * sqrtT;
    // > Reduced-precision
    sqrtT_rp = sqrtf(T_rp);
    d1_rp = (__logf(S_rp / X_rp) + (R_rp + 0.5f * V_rp * V_rp) * T_rp) / (V_rp * sqrtT_rp);
    d2_rp = d1_rp - V_rp * sqrtT_rp;

    // > Full-precision
    CNDD1 = cndGPU(d1);
    CNDD2 = cndGPU(d2);
    // > Reduced-precision
    CNDD1_rp = cndGPU(d1_rp);
    CNDD2_rp = cndGPU(d2_rp);

    // > Full-precision
    expRT = exp(- R * T);
    CallResult = S * CNDD1 - X * expRT * CNDD2;
    // > Reduced-precision
    expRT_rp = __expf(- R_rp * T_rp);
    CallResult_rp = S_rp * CNDD1_rp - X_rp * expRT_rp * CNDD2_rp;

    // > Full-precision
    PutResult  = X * expRT * (1.0f - CNDD2) - S * (1.0f - CNDD1);
    // > Reduced-precision
    PutResult_rp  = X_rp * expRT_rp * (1.0f - CNDD2_rp) - S_rp * (1.0f - CNDD1_rp);

    checkErrors(CallResult, CallResult_rp, CALL_RESULT_REL_ERR_THRESHOLD);
    checkErrors(PutResult, PutResult_rp, PUT_RESULT_REL_ERR_THRESHOLD);
}

__global__ void BlackScholesKernel(double *CallResult, double *PutResult, float *CallResult_rp, float *PutResult_rp,
        double *StockPrice, double *OptionStrike, double *OptionYears, double Riskfree, double Volatility, int optN) {

    const int opt = blockDim.x * blockIdx.x + threadIdx.x;
    if (opt < optN) {
        BlackScholesBodyGPU(CallResult[opt], PutResult[opt], CallResult_rp[opt], PutResult_rp[opt],
                StockPrice[opt], OptionStrike[opt], OptionYears[opt], Riskfree, Volatility);
    }
}

void BlackScholesGPU(double *CallResult, double *PutResult, float *CallResult_rp, float *PutResult_rp,
        double *StockPrice, double *OptionStrike, double *OptionYears, int optN) {
    BlackScholesKernel<<<DIV_UP(optN, BLOCK_SIZE), BLOCK_SIZE>>>(CallResult, PutResult, CallResult_rp,
            PutResult_rp, StockPrice, OptionStrike, OptionYears, RISKFREE, VOLATILITY, optN);
}




